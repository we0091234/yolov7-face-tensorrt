#include "hip/hip_runtime.h"
#include "postprocess.h"
#define MAX_LANDMARK 20
    static __device__ void affine_project(float* matrix, float x, float y, float* ox, float* oy){
        *ox = matrix[0] * x + matrix[1] * y + matrix[2];
        *oy = matrix[3] * x + matrix[4] * y + matrix[5];
    }

    static __global__ void decode_kernel(float* predict,int NUM_BOX_ELEMENT, int num_bboxes, int num_classes,int ckpt, float confidence_threshold, float* invert_affine_matrix, float* parray, int max_objects){  

        int position = blockDim.x * blockIdx.x + threadIdx.x;
		if (position >= num_bboxes) return;

        float* pitem     = predict + (5 + num_classes+ckpt*3) * position;
        float objectness = pitem[4];
        if(objectness < confidence_threshold)
            return;

        float* class_confidence = pitem + 5;
        float confidence        = *class_confidence++;
        int label               = 0;
        for(int i = 1; i < num_classes; ++i, ++class_confidence){
            if(*class_confidence > confidence){
                confidence = *class_confidence;
                label      = i;
            }
        }

        confidence *= objectness;
        if(confidence < confidence_threshold)
            return;
   
        int index = atomicAdd(parray, 1);
        if(index >= max_objects)
            return;
        // printf("index %d max_objects %d\n", index,max_objects);
        float cx         = pitem[0];
        float cy         = pitem[1];
        float width      = pitem[2];
        float height     = pitem[3];
        
        //五个关键点
        float *landmarks = pitem+5+num_classes;
        float landmark_array[MAX_LANDMARK*2];
        for (int i = 0; i<ckpt; i++)
        {
            landmark_array[2*i]=landmarks[3*i];
            landmark_array[2*i+1]=landmarks[3*i+1];
        }
     
        float left   = cx - width * 0.5f;
        float top    = cy - height * 0.5f;
        float right  = cx + width * 0.5f;
        float bottom = cy + height * 0.5f;

        affine_project(invert_affine_matrix, left,  top,    &left,  &top);
        affine_project(invert_affine_matrix, right, bottom, &right, &bottom);

        for(int i = 0; i<ckpt; i++)
        {
           affine_project(invert_affine_matrix, landmark_array[2*i],landmark_array[2*i+1],&landmark_array[2*i],&landmark_array[2*i+1]); 
        }
        // affine_project(invert_affine_matrix, x5,y5,&x5,&y5);

        float* pout_item = parray + 1 + index * NUM_BOX_ELEMENT;
        *pout_item++ = left;
        *pout_item++ = top;
        *pout_item++ = right;
        *pout_item++ = bottom;
        *pout_item++ = confidence;
        *pout_item++ = label;
        *pout_item++ = 1; // 1 = keep, 0 = ignore
        
     
        for(int i = 0; i<ckpt; i++)
        {
            *pout_item++=landmark_array[2*i];
            *pout_item++=landmark_array[2*i+1];
        }


    }

    static __device__ float box_iou(
        float aleft, float atop, float aright, float abottom, 
        float bleft, float btop, float bright, float bbottom
    ){

        float cleft 	= max(aleft, bleft);
        float ctop 		= max(atop, btop);
        float cright 	= min(aright, bright);
        float cbottom 	= min(abottom, bbottom);
        
        float c_area = max(cright - cleft, 0.0f) * max(cbottom - ctop, 0.0f);
        if(c_area == 0.0f)
            return 0.0f;
        
        float a_area = max(0.0f, aright - aleft) * max(0.0f, abottom - atop);
        float b_area = max(0.0f, bright - bleft) * max(0.0f, bbottom - btop);
        return c_area / (a_area + b_area - c_area);
    }

    static __global__ void nms_kernel(float* bboxes, int max_objects, float threshold,int NUM_BOX_ELEMENT){

        int position = (blockDim.x * blockIdx.x + threadIdx.x);
        int count = min((int)*bboxes, max_objects);
        if (position >= count) 
            return;
        
        // left, top, right, bottom, confidence, class, keepflag
        float* pcurrent = bboxes + 1 + position * NUM_BOX_ELEMENT;
        for(int i = 0; i < count; ++i){
            float* pitem = bboxes + 1 + i * NUM_BOX_ELEMENT;
            if(i == position || pcurrent[5] != pitem[5]) continue;

            if(pitem[4] >= pcurrent[4]){
                if(pitem[4] == pcurrent[4] && i < position)
                    continue;

                float iou = box_iou(
                    pcurrent[0], pcurrent[1], pcurrent[2], pcurrent[3],
                    pitem[0],    pitem[1],    pitem[2],    pitem[3]
                );

                if(iou > threshold){
                    pcurrent[6] = 0;  // 1=keep, 0=ignore
                    return;
                }
            }
        }
    } 

    void decode_kernel_invoker(float* predict, int  NUM_BOX_ELEMENT,int num_bboxes,int num_classes,int ckpt, float confidence_threshold, float* invert_affine_matrix, float* parray, int max_objects, hipStream_t stream)
    {
        int block = 256;
        int  grid =  ceil(num_bboxes / (float)block);
        
        decode_kernel<<<grid, block, 0, stream>>>(predict,NUM_BOX_ELEMENT, num_bboxes, num_classes,ckpt, confidence_threshold, invert_affine_matrix, parray, max_objects);
    }

    void nms_kernel_invoker(float* parray, float nms_threshold, int max_objects, hipStream_t stream,int NUM_BOX_ELEMENT){
        
        
        int block = max_objects<256? max_objects:256;
        int grid = ceil(max_objects / (float)block);
        nms_kernel<<<grid, block, 0, stream>>>(parray, max_objects, nms_threshold,NUM_BOX_ELEMENT);
    }
