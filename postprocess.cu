#include "hip/hip_runtime.h"
#include "postprocess.h"

    const int NUM_BOX_ELEMENT = 17;      // left, top, right, bottom, confidence, class, keepflag, 5 keypoints 
    static __device__ void affine_project(float* matrix, float x, float y, float* ox, float* oy){
        *ox = matrix[0] * x + matrix[1] * y + matrix[2];
        *oy = matrix[3] * x + matrix[4] * y + matrix[5];
    }

    static __global__ void decode_kernel(float* predict, int num_bboxes, int num_classes,int ckpt, float confidence_threshold, float* invert_affine_matrix, float* parray, int max_objects){  

        int position = blockDim.x * blockIdx.x + threadIdx.x;
		if (position >= num_bboxes) return;

        float* pitem     = predict + (5 + num_classes+ckpt*3) * position;
        float objectness = pitem[4];
        if(objectness < confidence_threshold)
            return;

        float* class_confidence = pitem + 5;
        float confidence        = *class_confidence++;
        int label               = 0;
        for(int i = 1; i < num_classes; ++i, ++class_confidence){
            if(*class_confidence > confidence){
                confidence = *class_confidence;
                label      = i;
            }
        }

        confidence *= objectness;
        if(confidence < confidence_threshold)
            return;
   
        int index = atomicAdd(parray, 1);
        if(index >= max_objects)
            return;
        // printf("index %d max_objects %d\n", index,max_objects);
        float cx         = pitem[0];
        float cy         = pitem[1];
        float width      = pitem[2];
        float height     = pitem[3];
        
        //五个关键点
        float *landmarks = pitem+5+num_classes;
        float x1         = landmarks[0];
        float y1         = landmarks[1];
        float x2         = landmarks[3];
        float y2         = landmarks[4];
        float x3         = landmarks[6];
        float y3         = landmarks[7];
        float x4         = landmarks[9];
        float y4         = landmarks[10];
        float x5         = landmarks[12];
        float y5         = landmarks[13];


        float left   = cx - width * 0.5f;
        float top    = cy - height * 0.5f;
        float right  = cx + width * 0.5f;
        float bottom = cy + height * 0.5f;

        affine_project(invert_affine_matrix, left,  top,    &left,  &top);
        affine_project(invert_affine_matrix, right, bottom, &right, &bottom);

        affine_project(invert_affine_matrix, x1,y1,&x1,&y1);
        affine_project(invert_affine_matrix, x2,y2,&x2,&y2);
        affine_project(invert_affine_matrix, x3,y3,&x3,&y3);
        affine_project(invert_affine_matrix, x4,y4,&x4,&y4);
        affine_project(invert_affine_matrix, x5,y5,&x5,&y5);

        float* pout_item = parray + 1 + index * NUM_BOX_ELEMENT;
        *pout_item++ = left;
        *pout_item++ = top;
        *pout_item++ = right;
        *pout_item++ = bottom;
        *pout_item++ = confidence;
        *pout_item++ = label;
        *pout_item++ = 1; // 1 = keep, 0 = ignore
        
         //five keypoint
        *pout_item++=x1;
        *pout_item++=y1;

        *pout_item++=x2;
        *pout_item++=y2;

        *pout_item++=x3;
        *pout_item++=y3;

        *pout_item++=x4;
        *pout_item++=y4;

        *pout_item++=x5;
        *pout_item++=y5;


    }

    static __device__ float box_iou(
        float aleft, float atop, float aright, float abottom, 
        float bleft, float btop, float bright, float bbottom
    ){

        float cleft 	= max(aleft, bleft);
        float ctop 		= max(atop, btop);
        float cright 	= min(aright, bright);
        float cbottom 	= min(abottom, bbottom);
        
        float c_area = max(cright - cleft, 0.0f) * max(cbottom - ctop, 0.0f);
        if(c_area == 0.0f)
            return 0.0f;
        
        float a_area = max(0.0f, aright - aleft) * max(0.0f, abottom - atop);
        float b_area = max(0.0f, bright - bleft) * max(0.0f, bbottom - btop);
        return c_area / (a_area + b_area - c_area);
    }

    static __global__ void nms_kernel(float* bboxes, int max_objects, float threshold){

        int position = (blockDim.x * blockIdx.x + threadIdx.x);
        int count = min((int)*bboxes, max_objects);
        if (position >= count) 
            return;
        
        // left, top, right, bottom, confidence, class, keepflag
        float* pcurrent = bboxes + 1 + position * NUM_BOX_ELEMENT;
        for(int i = 0; i < count; ++i){
            float* pitem = bboxes + 1 + i * NUM_BOX_ELEMENT;
            if(i == position || pcurrent[5] != pitem[5]) continue;

            if(pitem[4] >= pcurrent[4]){
                if(pitem[4] == pcurrent[4] && i < position)
                    continue;

                float iou = box_iou(
                    pcurrent[0], pcurrent[1], pcurrent[2], pcurrent[3],
                    pitem[0],    pitem[1],    pitem[2],    pitem[3]
                );

                if(iou > threshold){
                    pcurrent[6] = 0;  // 1=keep, 0=ignore
                    return;
                }
            }
        }
    } 

    void decode_kernel_invoker(float* predict, int num_bboxes, int num_classes,int ckpt, float confidence_threshold, float* invert_affine_matrix, float* parray, int max_objects, hipStream_t stream)
    {
        int block = 256;
        int  grid =  ceil(num_bboxes / (float)block);
        
        decode_kernel<<<grid, block, 0, stream>>>(predict, num_bboxes, num_classes,ckpt, confidence_threshold, invert_affine_matrix, parray, max_objects);
    }

    void nms_kernel_invoker(float* parray, float nms_threshold, int max_objects, hipStream_t stream){
        
        
        int block = max_objects<256? max_objects:256;
        int grid = ceil(max_objects / (float)block);
        nms_kernel<<<grid, block, 0, stream>>>(parray, max_objects, nms_threshold);
    }
